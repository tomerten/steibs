#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cmath>
#include <math.h>
#include <string>
#include <vector>
#include <set>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <boost/lexical_cast.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

#include <iterator>
#include <sstream>
#include <fstream>
#include <algorithm>

// load double 6 datastructure
#include "STE_DataStructures_double.cuh"

// load tfsTableData datastructure
#include "STE_TFS_double.cuh"

// load Hamiltonian functions
#include "STE_Longitudinal_Hamiltonian_double.cuh"

// constructor
STE_Longitudinal_Hamiltonian::STE_Longitudinal_Hamiltonian( hamiltonianParameters& params )
{
	
}

__host__ __device__ double STE_Longitudinal_Hamiltonian::tcoeff( hamiltonianParameters& params )
{
	return  (params.angularFrequency * params.eta * params.harmonicNumbers.x);
}

__host__ __device__ double STE_Longitudinal_Hamiltonian::pcoeff( hamiltonianParameters& params, double voltage ){
	return  (params.angularFrequency * voltage * params.particleCharge) / (2 * CUDA_PI_F * params.p0 * params.betar);
};

__host__ __device__ double STE_Longitudinal_Hamiltonian::HamiltonianTripleRf( hamiltonianParameters& params ){

	double kinetic, potential1, potential2, potential3;

	kinetic = 0.5 * tcoeff( params ) * pow(params.delta,2);

	double pcoeff1 = pcoeff( params , params.voltages.x );
	double pcoeff2 = pcoeff( params , params.voltages.y );
	double pcoeff3 = pcoeff( params , params.voltages.z );

	double phi1 = params.harmonicNumbers.x * params.angularFrequency * params.t;
	double phi2 = params.harmonicNumbers.y * params.angularFrequency * params.t;
	double phi3 = params.harmonicNumbers.z * params.angularFrequency * params.t;

	double h0oh1 = params.harmonicNumbers.x / params.harmonicNumbers.y;
	double h0oh2 = params.harmonicNumbers.x / params.harmonicNumbers.z;

	double h1oh0 = params.harmonicNumbers.y / params.harmonicNumbers.x;
	double h2oh0 = params.harmonicNumbers.z / params.harmonicNumbers.x;

	potential1 = pcoeff1 * (cos(phi1) - cos(params.phis) + (phi1 - params.phis) * sin(params.phis));
	potential2 = pcoeff2 * h0oh1 * (cos(phi2) - cos(h1oh0 * params.phis ) + (phi2 - h1oh0 * params.phis ) * sin(h1oh0 * params.phis));
	potential3 = pcoeff3 * h0oh2 * (cos(phi3) - cos(h2oh0 * params.phis) + (phi3 - h2oh0 * params.phis) * sin(h2oh0 * params.phis));

	return kinetic + potential1 + potential2 + potential3;
};