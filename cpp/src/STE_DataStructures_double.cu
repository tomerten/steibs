#include "hip/hip_runtime.h"
// random generator includes
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/xor_combine_engine.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <map>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <math.h>
#include <string>
#include <boost/math/tools/roots.hpp>
#include <thrust/tuple.h>

#include "STE_DataStructures_double.cuh"

#include <vector>

// to write 6-vector to screen
// __host__ std::ostream& operator<< (std::ostream& os, const float6& p)
// {
// 	os << std::setw(15) << "x" << std::setw(15) << "y" << std::setw(15) << "z" << std::endl;
// 	os << std::setw(15) << p.x << std::setw(15) << p.px << std::setw(15) << p.y << std::endl;
// 	os << std::setw(15) << p.py << std::setw(15) << p.t <<std::setw(15) << p.delta << std::endl;;
// 	return os;
// };

// // to write 2-vector to screen
// __host__ std::ostream& operator<< (std::ostream& os, const float2& p)
// {
// 	os << std::setw(21) << p.x << std::setw(21) << p.y;
// 	// os << printf("%.16f",p.x) << "\t" << printf("%.16f",p.y) << std::endl;
// 	// os << printf("%.16f \t %.16f\n",p.x,p.y);
// 	return os;
// };
